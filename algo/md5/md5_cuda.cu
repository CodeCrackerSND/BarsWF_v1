#include "hip/hip_runtime.h"
#include "md5.h"
#include "../../global_c.h"
#include "../../global.h"

//int md5_const_host[64] = {0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee, 0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501, 0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be, 0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821, 0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa, 0xd62f105d, 0x2441453, 0xd8a1e681, 0xe7d3fbc8, 0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed, 0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a, 0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c, 0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70, 0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x4881d05, 0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665, 0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039, 0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1, 0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1, 0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391};

//__constant__ int md5_const[64];// = {0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee, 0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501, 0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be, 0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821, 0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa, 0xd62f105d, 0x2441453, 0xd8a1e681, 0xe7d3fbc8, 0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed, 0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a, 0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c, 0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70, 0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x4881d05, 0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665, 0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039, 0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1, 0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1, 0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391};

__constant__ int target[4];// = {0xb182b498, 0xf4d2ac41, 0x1f636569, 0xaf4caf00};
__constant__ int code_len[1];// = {0xb182b498, 0xf4d2ac41, 0x1f636569, 0xaf4caf00};
__constant__ char charset_c[128];// = {0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee, 0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501, 0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be, 0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821, 0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa, 0xd62f105d, 0x2441453, 0xd8a1e681, 0xe7d3fbc8, 0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed, 0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a, 0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c, 0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70, 0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x4881d05, 0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665, 0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039, 0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1, 0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1, 0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391};

__device__ unsigned int *result_d;  //h - host, d - device
__device__ unsigned int *data_d;  //h - host, d - device

__global__ void md5_gpu_bruteforce_thread(unsigned int *data_d, unsigned int *result_d, unsigned int pwd_len, unsigned int gpu_len, int charset_len)
{
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int data[4];
	int a,b,c,d;
	const int len = pwd_len*8;//code_len[0];//
	for(int i=0;i<4;i++)data[i]=data_d[ix*4+i];//
	result_d[ix] = KEY_NOT_FOUND;
	//		if(s3_limit==26)data[0]=(data[0]&0xff00ffff)+(('a'+s3)<<16);//only if 3 symbols to brute force
	int ta,tb,tc,td;
	ta = target[0];
	tb = target[1];
	tc = target[2];
	td = target[3];
	unrollII (tc, td, ta, tb, data[2], S43, 0x2ad7d2bb); /* 63 */
	unrollII (td, ta, tb, tc, 0, S42, 0xbd3af235); /* 62 */
	unrollII (ta, tb, tc, td, 0, S41, 0xf7537e82); /* 61 */
	unrollII (tb, tc, td, ta, 0, S44, 0x4e0811a1); /* 60 */
	unrollII (tc, td, ta, tb, 0, S43, 0xa3014314); /* 59 */
	unrollII (td, ta, tb, tc, 0, S42, 0xfe2ce6e0); /* 58 */
	unrollII (ta, tb, tc, td, 0, S41, 0x6fa87e4f); /* 57 */
	unrollII (tb, tc, td, ta, data[1], S44, 0x85845dd1); /* 56 */
	unrollII (tc, td, ta, tb, 0, S43, 0xffeff47d); /* 55 */
	unrollII (td, ta, tb, tc, data[3], S42, 0x8f0ccc92); /* 54 */
	unrollII (ta, tb, tc, td, 0, S41, 0x655b59c3); /* 53 */
	unrollII (tb, tc, td, ta, 0, S44, 0xfc93a039); /* 52 */
	unrollII (tc, td, ta, tb, len, S43, 0xab9423a7); /* 51 */
	unrollII (td, ta, tb, tc, 0, S42, 0x432aff97); /* 50 */


	for(int s2=0;s2<charset_len;s2++)
	{
		data[0]=(data[0]&0xffff00ff)+((charset_c[s2])<<8);
		for(int s1=0;s1<charset_len;s1++)
		{
			data[0]=(data[0]&0xffffff00)+(charset_c[s1]);

			a=0x67452301;b=0xefcdab89;c=0x98badcfe;d=0x10325476;

			/* Round 1 */
			FF2 (a, b, c, d, data[0], S11, 0xd76aa478,0,0); /* 1 */
			FF2 (d, a, b, c, data[1], S12, 0xe8c7b756,1,1); /* 2 */
			FF2 (c, d, a, b, data[2], S13, 0x242070db,2,2); /* 3 */
			FF2 (b, c, d, a, data[3], S14, 0xc1bdceee,3,3); /* 4 */


			FF2 (a, b, c, d, 0, S11, 0xf57c0faf,4,15); /* 5 */ //5th and so forth is 0
			FF2 (d, a, b, c, 0, S12, 0x4787c62a,5,15); /* 6 */
			FF2 (c, d, a, b, 0, S13, 0xa8304613,6,15); /* 7 */
			FF2 (b, c, d, a, 0, S14, 0xfd469501,7,15); /* 8 */
			FF2 (a, b, c, d, 0, S11, 0x698098d8,8,15); /* 9 */
			FF2 (d, a, b, c, 0, S12, 0x8b44f7af,9,15); /* 10 */
			FF2 (c, d, a, b, 0, S13, 0xffff5bb1,10,15); /* 11 */
			FF2 (b, c, d, a, 0, S14, 0x895cd7be,11,15); /* 12 */
			FF2 (a, b, c, d, 0, S11, 0x6b901122,12,15); /* 13 */
			FF2 (d, a, b, c, 0, S12, 0xfd987193,13,15); /* 14 */
			FF2 (c, d, a, b, len, S13, 0xa679438e,14,14); /* 15 */
			FF2 (b, c, d, a, 0, S14, 0x49b40821,15,15); /* 16 */

			/* Round 2 */
			GG2 (a, b, c, d, data[1], S21, 0xf61e2562,16,1); /* 17 */
			GG2 (d, a, b, c, 0, S22, 0xc040b340,17,15); /* 18 */
			GG2 (c, d, a, b, 0, S23, 0x265e5a51,18,15); /* 19 */
			GG2 (b, c, d, a, data[0], S24, 0xe9b6c7aa,19,0); /* 20 */
			GG2 (a, b, c, d, 0, S21, 0xd62f105d,20,15); /* 21 */
			GG2 (d, a, b, c, 0, S22,  0x2441453,21,15); /* 22 */
			GG2 (c, d, a, b, 0, S23, 0xd8a1e681,22,15); /* 23 */
			GG2 (b, c, d, a, 0, S24, 0xe7d3fbc8,23,15); /* 24 */
			GG2 (a, b, c, d, 0, S21, 0x21e1cde6,24,15); /* 25 */
			GG2 (d, a, b, c, len, S22, 0xc33707d6,25,14); /* 26 */
			GG2 (c, d, a, b, data[3], S23, 0xf4d50d87,26,3); /* 27 */
			GG2 (b, c, d, a, 0, S24, 0x455a14ed,27,15); /* 28 */
			GG2 (a, b, c, d, 0, S21, 0xa9e3e905,28,15); /* 29 */
			GG2 (d, a, b, c, data[2], S22, 0xfcefa3f8,29,2); /* 30 */
			GG2 (c, d, a, b, 0, S23, 0x676f02d9,30,15); /* 31 */
			GG2 (b, c, d, a, 0, S24, 0x8d2a4c8a,31,15); /* 32 */

			/* Round 3 */
			HH2 (a, b, c, d, 0, S31, 0xfffa3942,32,15); /* 33 */
			HH2 (d, a, b, c, 0, S32, 0x8771f681,33,15); /* 34 */
			HH2 (c, d, a, b, 0, S33, 0x6d9d6122,34,15); /* 35 */
			HH2 (b, c, d, a, len, S34, 0xfde5380c,35,14); /* 36 */
			HH2 (a, b, c, d, data[1], S31, 0xa4beea44,36,1); /* 37 */
			HH2 (d, a, b, c, 0, S32, 0x4bdecfa9,37,15); /* 38 */
			HH2 (c, d, a, b, 0, S33, 0xf6bb4b60,38,15); /* 39 */
			HH2 (b, c, d, a, 0, S34, 0xbebfbc70,39,15); /* 40 */
			HH2 (a, b, c, d, 0, S31, 0x289b7ec6,40,15); /* 41 */
			HH2 (d, a, b, c, data[0], S32, 0xeaa127fa,41,0); /* 42 */
			HH2 (c, d, a, b, data[3], S33, 0xd4ef3085,42,3); /* 43 */
			HH2 (b, c, d, a, 0, S34,  0x4881d05,43,15); /* 44 */
			HH2 (a, b, c, d, 0, S31, 0xd9d4d039,44,15); /* 45 */
			HH2 (d, a, b, c, 0, S32, 0xe6db99e5,45,15); /* 46 */
			if(d!=td)continue;
			HH2 (c, d, a, b, 0, S33, 0x1fa27cf8,46,15); /* 47 */
			HH2 (b, c, d, a, data[2], S34, 0xc4ac5665,47,2); /* 48 */

			/* Round 4 */
			II2 (a, b, c, d, data[0], S41, 0xf4292244,48,0); /* 49 */

			if(a==ta && b==tb && c==tc)
			{
				result_d[ix] = KEY_FOUND;
			};
		}
	}
}

void cuda_get_mem(int device_id)
{
	//get memory for data transfers
	hipMalloc((void **) &data_d, sizeof(int)*4*gpu[device_id].thread_n*gpu[device_id].grid_n);
	hipMalloc((void **) &common_d, sizeof(md5_data));   

	hipHostMalloc((void**)&gpu[device_id].data_h, sizeof(int)*4*gpu[device_id].thread_n*gpu[device_id].grid_n);
	hipHostMalloc((void**)&gpu[device_id].common_h, sizeof(md5_data));
}

void init_md5_cuda(int device_id)
{
	hipMemcpyToSymbol(HIP_SYMBOL(target), hash_i, sizeof(hash_i));
	gpu[device_id].common_h->res[0] = -1;//key not found

	hipMemcpy(common_d, gpu[device_id].common_h, sizeof(md5_data), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(charset_c), perm::charset, perm::charset_len+1);
	//	hipMemcpyToSymbol(HIP_SYMBOL(md5_const), md5_const_host, sizeof(md5_const));
}

void cuda_free_mem(int device_id)
{
	hipFree(data_d);  
	hipFree(common_d);  

	free(gpu[device_id].data_h); 
	free(gpu[device_id].common_h); 
}
// The host CPU Mandebrot thread spawner
void do_123(const int device_id)
{
	dim3 threads(gpu[device_id].thread_n);
	dim3 grid(gpu[device_id].grid_n);

	//copy combinations to device
	hipEvent_t stop;
	__int64 tmp;
	getTimeDelta(tmp);

	CUDA_SAFE_CALL( hipEventCreate(&stop)  );

	hipMemcpyAsync(data_d, gpu[device_id].data_h, sizeof(int)*4*gpu[device_id].thread_n*gpu[device_id].grid_n, hipMemcpyHostToDevice,0);  

	md5_gpu_bruteforce_thread<<<grid, threads>>>(data_d, common_d, perm::pwd_len, perm::gpu_len, perm::charset_len);

	hipMemcpyAsync(gpu[device_id].common_h, common_d, sizeof(md5_data), hipMemcpyDeviceToHost, NULL);  
	hipError_t result = hipGetLastError();
	if(result != hipSuccess)
	{
		char msg[512];msg[0]=0;
		strcpy_s(msg, 512, "CUDA kernel error: ");
		strcat_s(msg, 512, hipGetErrorString(result));
		print_status(msg);
		is_finished = true;
		is_error_detected = true;
	}

	hipEventRecord(stop, 0);
	while( hipEventQuery(stop) == hipErrorNotReady )
	{
		Sleep(3);
	}
	CUDA_SAFE_CALL( hipEventDestroy(stop));

	if(gpu[device_id].common_h->res[0]!=-1)
	{
		is_finished = true;
		is_key_found = true;
		final_len = perm::pwd_len*8;
		for(int i=0;i<4;i++)final_key[i]=gpu[device_id].common_h->res[i];
	}
}
